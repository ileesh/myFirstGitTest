#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>

#define MAX_SHAREDSIZE	2048

__global__ void LoadStoreViaSharedMemory(int *In, int *Out)
{
#if 1
	int LoadStoreSize = MAX_SHAREDSIZE/blockDim.x;
	int beginIndex = threadIdx.x * LoadStoreSize;
	int endIndex = beginIndex + LoadStoreSize;

	// ���� �޸� �Ҵ�
	__shared__ int SharedMemory[MAX_SHAREDSIZE];
	int i;

	for(i = beginIndex; i < endIndex; i++)
		SharedMemory[i] = In[i];

	__syncthreads();

	for(i = beginIndex; i < endIndex; i++)
		Out[i] = SharedMemory[i];

	__syncthreads();
#else
	__shared__ int SharedMemory[MAX_SHAREDSIZE];

	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	SharedMemory[idx] = In[idx];
	Out[idx] = SharedMemory[idx];
#endif
}

int main()
{
	int size = MAX_SHAREDSIZE;
	int BufferSize = size*sizeof(int);

	int *In, *Out;
	In = (int *)malloc(BufferSize);
	Out = (int *)malloc(BufferSize);

	int i = 0;

	for(i = 0; i < size; i++) {
		In[i] = i;
		Out[i] = 0;
	}

	int *devIn, *devOut;
	hipMalloc((void **)&devIn, BufferSize);
	hipMalloc((void **)&devOut, BufferSize);
	
	hipMemcpy(devIn, In, BufferSize, hipMemcpyHostToDevice);

	//LoadStoreViaSharedMemory<<<32, 64>>>(devIn, devOut);
	LoadStoreViaSharedMemory<<<1, 512>>>(devIn, devOut);

	hipMemcpy(Out, devOut, BufferSize, hipMemcpyDeviceToHost);

	for(i = 0; i < 5; i++) 
		printf("%04d\n", Out[i]);

	printf("......\n");
	for(i = size-5; i < size; i++)
		printf("%04d\n", Out[i]);

	hipFree(devIn);
	hipFree(devOut);

	free(In);
	free(Out);

	getch();
}

